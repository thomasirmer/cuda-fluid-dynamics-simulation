#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "cpu_anim.h"
#include "utils.h"
#include "defines.cuh"

#define SIM_WIDTH   512
#define SIM_HEIGHT  512
#define NUM_THREADS 32

#define BLOCKBREITE  NUM_THREADS
#define GESAMTBREITE SIM_WIDTH

//#define _DEBUG_

// KERNEL FUNCTIONS >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
__shared__ Vector dots[BLOCKBREITE + 2][BLOCKBREITE + 2];

__device__ void calculateNewValue(float* res) {
	int threadID = getGlobalThreadId();

	int x = threadIdx.x % 32 + 1;
	int y = threadIdx.y % 32 + 1;

	res[0] = dots[x][y][0];
	res[1] = dots[x][y][1];

	for (int yi = -1; yi < 2; yi++) {
		for (int xi = -1; xi < 2; xi++) {
			if (!(xi == 0 && yi == 0)) {
				res[0] += dots[x + xi][y + yi][0];
				res[1] += dots[x + xi][y + yi][1];
			}
		}
	}

	res[0] /= 9;
	res[1] /= 9;
}

__device__ void copyToSharedMem(float* inDots) {

	//dots[]((float*) inDots)+(y*GESAMTBREITE)x*sizeof(float)

	//dots[x1 + 1][y1 + 1][0] = inDots[y * GESAMTBREITE + x][0];
	//dots[x1 + 1][y1 + 1][1] = inDots[y * GESAMTBREITE + x][1];

	int threadID = getGlobalThreadId();

	float initalValue = 0.0f;

	// füll Randwerte des shared memory
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		for (int x = 0; x <= BLOCKBREITE + 1; x++) {
			dots[x][0][0] = initalValue;
			dots[x][0][1] = initalValue;
			dots[x][BLOCKBREITE + 1][0] = initalValue;
			dots[x][BLOCKBREITE + 1][1] = initalValue;
		}
		for (int y = 0; y <= BLOCKBREITE + 1; y++) {
			dots[0][y][0] = initalValue;
			dots[0][y][1] = initalValue;
			dots[BLOCKBREITE + 1][y][0] = initalValue;
			dots[BLOCKBREITE + 1][y][1] = initalValue;
		}
	}

	__syncthreads();

	int x = threadIdx.x % 32 + 1;
	int y = threadIdx.y % 32 + 1;

	dots[x][x][0] = inDots[threadID * 2];
	dots[y][y][1] = inDots[threadID * 2 + 1];

	// copy to shared memory (inefficient)
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		for (int i = 0; i < gridDim.x + 2; i++) {
			dots[i][0][0] = inDots[threadID * 2];
			dots[i][0][1] = inDots[threadID * 2 + 1];
		}
		for (int j = 0; j < gridDim.y + 2; j++) {

		}
	}

	// ab hier große scheiße

	int offsetAboveX = (threadID * 2) - (gridDim.x * blockDim.x) * 2;
	int offsetAboveY = (threadID * 2 + 1) - (gridDim.x * blockDim.x) * 2;

	// Wir befinden uns nicht in der ersten Zeile
	if (blockIdx.y != 0) {
		if (threadIdx.y == 0) { // 1. Zeile der Threads kopiert x-Werte
			dots[x][0][0] = inDots[offsetAboveX];
		}
		if (threadIdx.y == 1) { // 2. Zeile der Threads kopiert y-Werte
			dots[x][0][1] = inDots[offsetAboveY];
		}
	}

	int offsetBelowX = (threadID * 2) + (gridDim.x * blockDim.x) * 2;
	int offsetBelowY = (threadID * 2 + 1) + (gridDim.x * blockDim.x) * 2;

	// Nicht in der letzten Zeile
	if (blockIdx.y != 512 / 32 - 1) {
		if (threadIdx.y == 2) {
			dots[x][33][0] = inDots[offsetAboveX];
		}
		if (threadIdx.y == 3) {
			dots[x][33][1] = inDots[offsetAboveY];
		}
	}

	// Nicht erste Spalte
	if (blockIdx.x != 0) {


	} else {

	}

	// Nicht letzte Spalte
	if (blockIdx.x == 512 / 32 - 1) {

	} else {

	}

	__syncthreads();
}

__global__ void simulate(float* inDots, float* outDots) {

	int threadID = getGlobalThreadId();

	copyToSharedMem(inDots);

	float res[2];
	calculateNewValue(res);
	outDots[threadID * 2] 		= res[0];
	outDots[threadID * 2 + 1] 	= res[1];
}

// global threadID for 2D grid of 2D blocks
__device__ int getGlobalThreadId() {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y)
			+ (threadIdx.y * blockDim.x) + threadIdx.x;

	return threadId;
}

// vector calculations
__device__ float getVectorLength(float xCoord, float yCoord) {
	return (float) sqrt(xCoord * xCoord + yCoord * yCoord);
}

// return the angle of the vector
__device__ float getVectorAngle(float xCoord, float yCoord) {
	float angleRad = atan2(yCoord, xCoord);
	float angleDeg = (angleRad / M_PI) * 180.0f;
	return angleDeg;
}

//// simulation function (copy input --> output)
//__global__ void simulate(float* inputValues, float* outputValues) {
//	int threadId = getGlobalThreadId();
//
//	if (threadId < SIM_WIDTH * SIM_HEIGHT) {
//		outputValues[threadId * 2] = inputValues[threadId * 2];
//		outputValues[threadId * 2 + 1] = inputValues[threadId * 2 + 1];
//	}
//}

__device__ void hsv2rgb(unsigned int hue, unsigned int sat, unsigned int val,
		unsigned char * r, unsigned char * g, unsigned char * b,
		unsigned char maxBrightness) {

	unsigned int H_accent = hue / 60;
	unsigned int bottom = ((255 - sat) * val) >> 8;
	unsigned int top = val;
	unsigned char rising = ((top - bottom) * (hue % 60)) / 60 + bottom;
	unsigned char falling = ((top - bottom) * (60 - hue % 60)) / 60 + bottom;

	switch (H_accent) {
	case 0:
		*r = top;
		*g = rising;
		*b = bottom;
		break;

	case 1:
		*r = falling;
		*g = top;
		*b = bottom;
		break;

	case 2:
		*r = bottom;
		*g = top;
		*b = rising;
		break;

	case 3:
		*r = bottom;
		*g = falling;
		*b = top;
		break;

	case 4:
		*r = rising;
		*g = bottom;
		*b = top;
		break;

	case 5:
		*r = top;
		*g = bottom;
		*b = falling;
		break;
	}
	// Scale values to maxBrightness
	*r = *r * maxBrightness / 255;
	*g = *g * maxBrightness / 255;
	*b = *b * maxBrightness / 255;
}
// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

void anim_exit(DataBlock *d) {
	hipFree(d->dev_inSrc);
	hipFree(d->dev_outSrc);
	hipFree(d->dev_constSrc);
	hipFree(d->bitmap);
	hipFree(d->output_bitmap);
}

void anim_gpu(DataBlock *d, int ticks) {
	// TODO: Animation
	dim3 blocks(ceil(SIM_WIDTH / NUM_THREADS), ceil(SIM_HEIGHT / NUM_THREADS));
	dim3 threads(NUM_THREADS, NUM_THREADS);
	CPUAnimBitmap* bitmap = d->bitmap;

	//copy_const_kernel<<<blocks, threads>>>(d->dev_inSrc, d->dev_constSrc);
	simulate<<<blocks, threads>>>(d->dev_inSrc, d->dev_outSrc);

	// TODO: Implement float_to_color that it uses both values!
	float_to_color<<<blocks, threads>>>(d->output_bitmap, d->dev_outSrc);

	swap(d->dev_inSrc, d->dev_outSrc);

	hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(),
			hipMemcpyDeviceToHost);
}

int main(void) {
	printf("Starting CUDA-Application - Parallel Fluid Simulation ...\n");

	// TODO: initialize all the stuff
	//Vector2D* vectorField = new Vector2D[SIM_HEIGHT * SIM_WIDTH];
	Vector* vectorField = new Vector[SIM_HEIGHT * SIM_WIDTH];

	// TODO: do simulation (ihno)

	// TODO: visualize (thomas)
	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
	DataBlock dataBlock;
	CPUAnimBitmap animBitmap(SIM_WIDTH, SIM_HEIGHT, &dataBlock);
	dataBlock.bitmap = &animBitmap;

	// allocate device memory
	int imageSize = animBitmap.image_size(); // image_size() returns width * height * 4
	hipMalloc((void**) &dataBlock.output_bitmap, imageSize * 2);
	hipMalloc((void**) &dataBlock.dev_inSrc, imageSize * 2);
	hipMalloc((void**) &dataBlock.dev_outSrc, imageSize * 2);

	// initialize vectorField
	for (int i = 0; i < SIM_HEIGHT * SIM_WIDTH; i++) {
		// random values between [0.0 ... 1.0]
		//float xValue = (float) rand() / (float) RAND_MAX;
		//float yValue = (float) rand() / (float) RAND_MAX;

		// circle values
		float xValue = cos(
				((float) i / (SIM_WIDTH * SIM_HEIGHT - 1)) * 2 * M_PI);
		float yValue = sin(
				((float) i / (SIM_WIDTH * SIM_HEIGHT - 1)) * 2 * M_PI);

		// assign to vectorField
		vectorField[i][0] = xValue;
		vectorField[i][1] = yValue;
	}

	// copy input values to device
	hipMemcpy(dataBlock.dev_inSrc, vectorField, imageSize * 2,
			hipMemcpyHostToDevice);

	// start simulation
	animBitmap.anim_and_exit((void (*)(void*, int)) anim_gpu, (void (*)(void*))anim_exit );
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
}
