
#include <stdio.h>
#include <stdlib.h>

#include "cpu_anim.h"
#include "defines.cuh"

#define SIM_WIDTH  32
#define SIM_HEIGHT 32

void anim_exit(DataBlock *d) {
	hipFree(d->dev_inSrc);
	hipFree(d->dev_outSrc);
	hipFree(d->dev_constSrc);
}

void anim_gpu(DataBlock *d, int ticks) {
//	dim3 blocks(DIM / 16, DIM / 16);
//	dim3 threads(16, 16);
//	CPUAnimBitmap *bitmap = d->bitmap;
//
//	// i calculations before updating the bitmap
//	for (int i = 0; i < 25; i++) {
//		copy_const_kernel<<<blocks, threads>>>(d->dev_inSrc, d->dev_constSrc);
//		blend_kernel<<<blocks, threads>>>(d->dev_outSrc, d->dev_inSrc);
//		swap(d->dev_inSrc, d->dev_outSrc);
//	}
//
//	float_to_color<<<blocks, threads>>>(d->output_bitmap, d->dev_inSrc);
//
//	hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(),
//			hipMemcpyDeviceToHost);

	// TODO: Animation

}

int main (void) {
	printf("Starting CUDA-Application - Parallel Fluid Simulation ...\n");

	// TODO: initialize all the stuff
	//Vector2D* vectorField = new Vector2D[SIM_HEIGHT * SIM_WIDTH];
	Vector* vectorField = new Vector[SIM_HEIGHT * SIM_WIDTH];

	// TODO: do simulation (ihno)

	// TODO: visualize (thomas)
	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>

	// random initialize vectorField
	for (int i = 0; i < SIM_HEIGHT * SIM_WIDTH; i++) {
		// random values between 0.0 - 1.0
		float xValue = (float) rand() / (float) RAND_MAX;
		float yValue = (float) rand() / (float) RAND_MAX;

		// assign to vectorField
		vectorField[i][0] = xValue;
		vectorField[i][1] = yValue;
	}

	DataBlock dataBlock;
	CPUAnimBitmap animBitmap(SIM_WIDTH, SIM_HEIGHT, &dataBlock);
	dataBlock.bitmap = &animBitmap;

	int imageSize = animBitmap.image_size();
	hipMalloc((void**) &dataBlock.dev_inSrc,    imageSize);
	hipMalloc((void**) &dataBlock.dev_outSrc,   imageSize);
	hipMalloc((void**) &dataBlock.dev_constSrc, imageSize);

	// initialize constant data (border pixels are all zero)
	// top border
	for (int i = 0; i < SIM_WIDTH; i++) {
		vectorField[i][0] = 0;
		vectorField[i][1] = 0;
	}
	// bottom border
	for (int i = SIM_WIDTH * SIM_HEIGHT - SIM_WIDTH; i < SIM_WIDTH * SIM_HEIGHT; i++) {
		vectorField[i][0] = 0;
		vectorField[i][1] = 0;
	}
	// left border
	for (int i = 0; i < SIM_WIDTH * SIM_HEIGHT; i += SIM_WIDTH) {
		vectorField[i][0] = 0;
		vectorField[i][1] = 0;
	}
	// right border
	for (int i = SIM_WIDTH; i < SIM_WIDTH * SIM_HEIGHT; i += SIM_WIDTH) {
		vectorField[i][0] = 0;
		vectorField[i][1] = 0;
	}

	hipMemcpy(dataBlock.dev_constSrc, vectorField, imageSize, hipMemcpyHostToDevice);
	hipMemcpy(dataBlock.dev_inSrc   , vectorField, imageSize, hipMemcpyHostToDevice);

	animBitmap.anim_and_exit((void (*)(void*,int)) anim_gpu,
                          	 (void (*)(void*))     anim_exit );

	// destruction
	delete[] vectorField;
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

	printf("...finished!\n");
}
