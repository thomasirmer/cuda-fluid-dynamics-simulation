#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <float.h>

#include <iostream>
#include <fstream>

#include "cpu_anim.cuh"
#include "utils.cuh"
#include "defines.cuh"
#include "kernel_functions.cuh"

// ****************************************************************************
// KERNEL FIELDS
// ****************************************************************************

__shared__ Vector sDots[BLOCK_WIDTH + 2][BLOCK_WIDTH + 2];

// ****************************************************************************
// KERNEL FUNCTIONS
// ****************************************************************************

// global threadID for 2D grid of 2D blocks
__device__ int getGlobalThreadId() {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

// offset for 1-dimensional array index
__device__ int getArrayOffset() {
	int xOffset = threadIdx.x + blockIdx.x * blockDim.x;
	int yOffset = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = (xOffset + yOffset * blockDim.x * gridDim.x) * 2;

	return offset;
}

// navier-stokes diffusion
__device__ void diffusion(float* newValues) {
	int tidx = threadIdx.x + 1; // thread x-coordinate inside block
	int tidy = threadIdx.y + 1; // thread y-coordinate inside block

	const float k = 0.5f; // kinematic viscosity
	const float dt = 1.0f; // timestep - for future use

	newValues[0] = k * dt
			* (sDots[tidx - 1][tidy][0] + sDots[tidx][tidy - 1][0] + sDots[tidx + 1][tidy][0] + sDots[tidx][tidy + 1][0]
					- 4 * sDots[tidx][tidy][0]);

	newValues[1] = k * dt
			* (sDots[tidx - 1][tidy][1] + sDots[tidx][tidy - 1][1] + sDots[tidx + 1][tidy][1] + sDots[tidx][tidy + 1][1]
					- 4 * sDots[tidx][tidy][1]);
}

// navier-stokes advection
__device__ void advection(float* newValues) {
	int tidx = threadIdx.x + 1; // thread x-coordinate inside block
	int tidy = threadIdx.y + 1; // thread y-coordinate inside block

	float dVelocityX = ((sDots[tidx - 1][tidy][0] - sDots[tidx + 1][tidy][0]) / 2)
			+ ((sDots[tidx][tidy - 1][0] - sDots[tidx][tidy + 1][0]) / 2);

	float dVelocityY = ((sDots[tidx - 1][tidy][1] - sDots[tidx + 1][tidy][1]) / 2)
			+ ((sDots[tidx][tidy - 1][1] - sDots[tidx][tidy + 1][1]) / 2);

	newValues[0] -= (sDots[tidx][tidy][0] * dVelocityX);
	newValues[1] -= (sDots[tidx][tidy][1] * dVelocityY);
}

__device__ float getWeightByAngle(float angle) {
	if (angle >= -180 && angle <= -90)
		return (angle + 90) / -90;
	if (angle >= -90 && angle <= 0)
		return (angle + 90) / 90;
	if (angle >= 0 && angle <= 90)
		return (angle - 90) / -90;
	if (angle >= 90 && angle <= 180)
		return (angle - 90) / 90;
	else
		return 0;
}

// simulate inside each block
__device__ void calculateNewValue(float* newValues) {

	int tidx = threadIdx.x + 1; // thread x-coordinate inside block
	int tidy = threadIdx.y + 1; // thread y-coordinate inside block

	if (blockIdx.y == 2 && blockIdx.x == 2) {
		newValues[0] = 80.0f;
		newValues[1] = 50.0f;
	} else if (blockIdx.y == 13 && blockIdx.x == 3) {
		newValues[0] = 60.0f;
		newValues[1] = -70.0f;
	} else if (blockIdx.y == 2 && blockIdx.x == 12) {
		newValues[0] = -30.0f;
		newValues[1] = 90.0f;
	} else if (blockIdx.y == 11 && blockIdx.x == 10) {
		newValues[0] = -70.0f;
		newValues[1] = -70.0f;
	} else {

		float weight = 1.0f;

		newValues[0] = weight * sDots[tidx][tidy][0];
		newValues[1] = weight * sDots[tidx][tidy][1];

		for (int x = -1; x <= 1; x++) {
			for (int y = -1; y <= 1; y++) {
				if (x != 0 && y != 0) {
					float neighborX = sDots[tidx + x][tidy + y][0];
					float neighborY = sDots[tidx + x][tidy + y][1];

					float angle = getAngleBetween(-x, -y, neighborX, neighborY);

					// add x-y-values of neighbor pixels
					float currentWeight = (1.5 - abs(angle) / 180.0f);
					newValues[0] += currentWeight * neighborX;
					newValues[1] += currentWeight * neighborY;
					weight += currentWeight;

					// rotate based on direction of neighbor pixels
					float relativeAngle = 0.0f;
					if (getVectorLength(neighborX, neighborY) >= 0.0f + FLT_EPSILON) {
						relativeAngle = getAngleBetween(newValues[0], newValues[1], neighborX, neighborY);
					}
					float rotationSpeed = 0.1f; //getVectorLength(sDots[tidx + x][tidy + y][0], sDots[tidx + x][tidy + y][1]);
					float rotationAngle = relativeAngle * rotationSpeed / 180.0f * M_PI;
					float cosinus = cos(rotationAngle);
					float sinus = sin(rotationAngle);
					newValues[0] = (newValues[0] * cosinus - newValues[1] * sinus);
					newValues[1] = (newValues[0] * sinus + newValues[1] * cosinus);
				}
			}
		}

		newValues[0] /= weight;
		newValues[1] /= weight;
	}
}

// copy input data to shared memory
__device__ void copyToSharedMem(float* inVelocityField) {

	int x = threadIdx.x;
	int y = threadIdx.y;

	float initalBorderValue = 0.0f;

	int xPos = blockIdx.x * blockDim.x * 2 + x;

	sDots[x / 2 + 1][y + 1][x % 2] = inVelocityField[(blockIdx.y * 32 + y) * GESAMTBREITE + xPos];
	sDots[x / 2 + 1 + 16][y + 1][x % 2] = inVelocityField[(blockIdx.y * 32 + y) * GESAMTBREITE + xPos + 32];

	// not in first row
	if (blockIdx.y != 0) {
		if (y == 0)
			sDots[x / 2 + 1][0][x % 2] = inVelocityField[(blockIdx.y * 32 - 1) * GESAMTBREITE + xPos];
		else if (y == 1)
			sDots[x / 2 + 1 + 16][0][x % 2] = inVelocityField[(blockIdx.y * 32 - 1) * GESAMTBREITE + xPos + 32];
		else if (y == 2 && (x == 0 || x == 1))
			sDots[0][0][(x + 1) % 2] =
					blockIdx.x != 0 ?
							inVelocityField[(blockIdx.y * 32 - 1) * GESAMTBREITE + xPos - (x * 2 + 1)] :
							initalBorderValue;
		else if (y == 3 && (x == 0 || x == 1))
			sDots[33][0][x] =
					blockIdx.x != (512 / 32 - 1) ?
							inVelocityField[(blockIdx.y * 32 - 1) * GESAMTBREITE + xPos + (32 * 2)] : initalBorderValue;
	} else {
		if (y == 0)
			sDots[x / 2 + 1][0][x % 2] = initalBorderValue;
		else if (y == 1)
			sDots[x / 2 + 1 + 16][0][x % 2] = initalBorderValue;
		else if (y == 2 && (x == 0 || x == 1))
			sDots[0][0][x] = initalBorderValue;
		else if (y == 3 && (x == 0 || x == 1))
			sDots[33][0][x] = initalBorderValue;
	}

	// not in last row (?) access violation (?)
	if (blockIdx.y != (512 / 32 - 1)) {
		if (y == 4)
			sDots[x / 2 + 1][33][x % 2] = inVelocityField[((blockIdx.y + 1) * 32) * GESAMTBREITE + xPos];
		else if (y == 5)
			sDots[x / 2 + 1 + 16][33][x % 2] = inVelocityField[((blockIdx.y + 1) * 32) * GESAMTBREITE + xPos + 32];
		else if (y == 6 && (x == 0 || x == 1))
			sDots[0][33][(x + 1) % 2] =
					blockIdx.x != 0 ?
							inVelocityField[((blockIdx.y + 1) * 32) * GESAMTBREITE + xPos - (x * 2 + 1)] :
							initalBorderValue;
		else if (y == 7 && (x == 0 || x == 1))
			sDots[33][33][x] =
					blockIdx.x != (512 / 32 - 1) ?
							inVelocityField[((blockIdx.y + 1) * 32) * GESAMTBREITE + xPos + (32 * 2)] :
							initalBorderValue;
	} else {
		if (y == 4)
			sDots[x + 1][33][0] = 0;
		else if (y == 5)
			sDots[x + 1][33][1] = 0;
		else if (y == 6 && (x == 0 || x == 1))
			sDots[0][33][x] = 0;
		else if (y == 7 && (x == 0 || x == 1))
			sDots[33][33][x] = 0;
	}

	// not in first column
	if (blockIdx.x != 0) {
		if (y == 8)
			sDots[0][x / 2 + 1][(x + 1) % 2] = inVelocityField[(blockIdx.y * 32) * GESAMTBREITE + (x / 2) * GESAMTBREITE
					+ blockIdx.x * 64 - (1 + x % 2)];
		if (y == 9)
			sDots[0][x / 2 + 1 + 16][(x + 1) % 2] = inVelocityField[(blockIdx.y * 32) * GESAMTBREITE
					+ (x / 2 + 16) * GESAMTBREITE + blockIdx.x * 64 - (1 + x % 2)];

	} else {
		if (y == 8)
			sDots[0][x + 1][0] = 0;
		if (y == 9)
			sDots[0][x + 1][1] = 0;
	}

	// not in last column
	if (blockIdx.x != (512 / 32 - 1)) {
		if (y == 10)
			sDots[33][x / 2 + 1][x % 2] = inVelocityField[(blockIdx.y * 32) * GESAMTBREITE + (x / 2) * GESAMTBREITE
					+ blockIdx.x * 64 + x % 2 + 64];
		if (y == 11)
			sDots[33][x / 2 + 1 + 16][x % 2] = inVelocityField[(blockIdx.y * 32) * GESAMTBREITE
					+ (x / 2 + 16) * GESAMTBREITE + blockIdx.x * 64 + x % 2 + 64];
	} else {
		if (y == 10)
			sDots[33][x + 1][0] = 0;
		if (y == 11)
			sDots[33][x + 1][1] = 0;
	}

	__syncthreads();
}

// simulation function (will be called once per run loop)
__global__ void simulate(float* inVelocityField, float* outVelocityField) {

	int offset = getArrayOffset();
	copyToSharedMem(inVelocityField);
	float newValues[2];
	calculateNewValue(newValues);
	outVelocityField[offset] = newValues[0];
	outVelocityField[offset + 1] = newValues[1];
}

// ****************************************************************************
// ANIMATION FUNCTIONS
// ****************************************************************************

// exit function for run-loop
void anim_exit(DataBlock *d) {
	hipFree(d->dev_inSrc);
	hipFree(d->dev_outSrc);
	hipFree(d->dev_constSrc);
	hipFree(d->bitmap);
	hipFree(d->output_bitmap);
}

// animation run-loop function
void anim_gpu(DataBlock *d, int ticks) {
	dim3 blocks(ceil(SIM_WIDTH / BLOCK_WIDTH), ceil(SIM_HEIGHT / BLOCK_WIDTH));
	dim3 threads(BLOCK_WIDTH, BLOCK_WIDTH);
	CPUAnimBitmap* bitmap = d->bitmap;

	simulate<<<blocks, threads>>>(d->dev_inSrc, d->dev_outSrc);
	float_to_color<<<blocks, threads>>>(d->output_bitmap, d->dev_outSrc);
	swap(d->dev_inSrc, d->dev_outSrc);
	hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost);
}

// ****************************************************************************
// MAIN
// ****************************************************************************

int main(void) {
	printf("Starting CUDA-Application - Parallel Fluid Simulation ...\n");

	// initialize data field which will be used for all further calculation
	Vector* velocityField = new Vector[SIM_HEIGHT * SIM_WIDTH];

	// set up stuff for graphical output
	DataBlock dataBlock;
	CPUAnimBitmap animBitmap(SIM_WIDTH, SIM_HEIGHT, &dataBlock);
	dataBlock.bitmap = &animBitmap;

	// allocate device memory
	int imageSize = animBitmap.image_size(); // image_size() returns width * height * 4
	hipMalloc((void**) &dataBlock.output_bitmap, imageSize * 2);
	hipMalloc((void**) &dataBlock.dev_inSrc, imageSize * 2);
	hipMalloc((void**) &dataBlock.dev_outSrc, imageSize * 2);

	// initialize vectorField
	for (int i = 0; i < SIM_HEIGHT * SIM_WIDTH; i++) {
		// circle values
//		float xValue = cos(((float) i / (SIM_WIDTH * SIM_HEIGHT - 1)) * 2 * M_PI);
//		float yValue = sin(((float) i / (SIM_WIDTH * SIM_HEIGHT - 1)) * 2 * M_PI);

		// assign to vectorField
		velocityField[i][0] = 0;
		velocityField[i][1] = 0;
	}

	// copy input values to device
	hipMemcpy(dataBlock.dev_inSrc, velocityField, imageSize * 2, hipMemcpyHostToDevice);

	// start simulation
	animBitmap.anim_and_exit((void (*)(void*, int)) anim_gpu, (void (*)(void*))anim_exit );
}
