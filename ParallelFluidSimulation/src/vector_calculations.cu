
#include <hip/hip_runtime.h>
#include <math.h>
#include <float.h>

// vector calculations
__device__ float getVectorLength(float xCoord, float yCoord) {
	return (float) sqrt(xCoord * xCoord + yCoord * yCoord);
}

// return the angle of the vector
__device__ float getVectorAngle(float xCoord, float yCoord) {
	float angleRad = atan2(yCoord, xCoord);
	float angleDeg = (angleRad / M_PI) * 180.0f;
	return angleDeg;
}

__device__ float getAngleBetween(float x1, float y1, float x2, float y2) {

	// normalize vectors
	float lenght1 = sqrt(x1 * x1 + y1 * y1);
	float normX1;
	float normY1;
	if (lenght1 <= 0.0f + FLT_EPSILON) {
		normX1 = 0;
		normY1 = 0;
	} else {
		normX1 = x1 / lenght1;
		normY1 = y1 / lenght1;
	}

	float lenght2 = sqrt(x2 * x2 + y2 * y2);
	float normX2;
	float normY2;
	if (lenght2 <= 0.0f + FLT_EPSILON) {
		normX2 = 0;
		normY2 = 0;
	} else {
		normX2 = x2 / lenght2;
		normY2 = y2 / lenght2;
	}

	// calculate angle
	float angle = (atan2(normY2, normX2) - atan2(normY1, normX1)) / M_PI * 180.0f;

	// correct angle at 180°-overflow
	if (angle < -180.0f)
		angle += 360.0f;
	if (angle > 180.0f)
		angle -= 360.0f;

	return angle;
}
