
#include <hip/hip_runtime.h>
// vector calculations
__device__ float getVectorLength(float xCoord, float yCoord) {
	return (float) sqrt(xCoord * xCoord + yCoord * yCoord);
}

// return the angle of the vector
__device__ float getVectorAngle(float xCoord, float yCoord) {
	float angleRad = atan2(yCoord, xCoord);
	float angleDeg = (angleRad / M_PI) * 180.0f;
	return angleDeg;
}
